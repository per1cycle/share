#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "common.cuh"


void usage()
{
    std::cout << "Usage: ./a.out <matrix dimension>\n"
                << "Notice, the a x b = c\n"
                << "a: N x M matrix\n"
                << "b: M x K matrix\n"
                << "c: N x K matrix\n"
                // << "to align with cublas, we use b^T x a^T = c^T\n"
                // << "aka: b K x M matrix, a M x N matrix, c K x N matrix\n"
                << "In the example here the N = M = K\n";
}

/**
 * assuming all of data can be divided by block
 * not fully understand why do this...
 */
template <const int BLK>
__global__ void sgemm_share_memory_caching(int N, int M, int K, float *a, float *b, float *c, float alpha, float beta)
{
    int c_row = blockIdx.x;
    int c_col = blockIdx.y;

    a += c_row * BLK * M;
    b += c_col * BLK;
    c += c_row * K * BLK + c_col * BLK;

    int thread_row = threadIdx.x / BLK;
    int thread_col = threadIdx.x % BLK;
    float tmp = 0.0f;

    __shared__ float a_share[BLK * BLK];
    __shared__ float b_share[BLK * BLK];

    for(int k = 0; k < M; k += BLK)
    {
        a_share[thread_row * BLK + thread_col] = a[thread_row * M + thread_col];
        b_share[thread_row * BLK + thread_col] = b[thread_row * K + thread_col];

        __syncthreads();

        a += BLK;
        b += BLK * K;

        for(int inner = 0; inner < BLK; inner ++)
        {
            tmp += a_share[thread_row * BLK + inner] * b_share[inner * BLK + thread_col];
        }

        __syncthreads();

    }

    c[thread_row * K + thread_col] = alpha * tmp + beta;
}
/**
 * Time:           0.63702ms.
 * GFlop:          137.456
 * GFLOPS:         215.779
 * Percentage:     4.9321%.
 */
int main(int argc, char ** argv)
{
    if(argc < 2)
    {
        usage();
        exit(1);
    }
    uint N;
    const int BLK = 32;

    N = atoi(argv[1]);

    size_t size = sizeof(float) * N * N;
    float flops = 1.0 * N * N * (2 * N + 1);
    
    float *h_a = new float[N * N];
    float *h_b = new float[N * N];
    float *h_c = new float[N * N];

    generate_float_matrix(h_a, N, N);
    generate_float_matrix(h_b, N, N);
    memset(h_c, 0, size);

    float *d_a;
    float *d_b;
    float *d_c;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);

    // fix typo.
    dim3 grid_dim = {N / BLK, N / BLK};
    dim3 blk_dim = {BLK * BLK};

    // start measuring.
    float elapsed; // in milisecond

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    sgemm_share_memory_caching<BLK><<<grid_dim, blk_dim>>>(N, N, N, d_a, d_b, d_c, 1.0f, 0.0f);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    // std::cout << "GFLOPS: " << flops / 1000000000.0f / elapsed * 1000.0f << std::endl;

    float gflops = flops / 1000000000.0f;
    elapsed = elapsed / 1000.0f; // to second

    std::cout << "Time: \t\t" << elapsed << "ms.\n"
            << "GFlop: \t\t" << gflops << "\n"
            << "GFLOPS: \t" << gflops / elapsed << "\n"
            << "Percentage: \t" << (gflops / elapsed) / 4591.26f * 100.0 << "%.\n";

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    return 0;
}