
#include <hip/hip_runtime.h>
#include <iostream>

const int N = 16;
__global__ void MatAdd(float A[N], float B[N], float C[N])
{
    printf("Execute in Block dim: (%d), blockIdx: (%d), thread: %d\n", blockDim.x, blockIdx.x, threadIdx.x);
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    std::cout << "Cuda works" << std::endl;
    float A[N], B[N], C[N];
    for (int i = 0; i < N; i++)
    {
        A[i] = i;
        B[i] = i;
    }
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N * sizeof(float), hipMemcpyHostToDevice);
    MatAdd<<<4, N / 4>>>(d_A, d_B, d_C);
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i ++)
    {
        std::cout << C[i] << ' ';
    }
    std::cout << std::endl;
    std::cout << "Cuda finished" << std::endl;
}