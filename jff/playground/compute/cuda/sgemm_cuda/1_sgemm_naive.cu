#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "common.cuh"

int N;
uint BLK = 32;

void usage()
{
    std::cout << "Usage: ./a.out <matrix dimension>\n"
                << "Notice, the a x b = c\n"
                << "a: N x M matrix\n"
                << "b: M x K matrix\n"
                << "c: N x K matrix\n"
                // << "to align with cublas, we use b^T x a^T = c^T\n"
                // << "aka: b K x M matrix, a M x N matrix, c K x N matrix\n"
                << "In the example here the N = M = K\n";
}

/**
 * assuming all of data can be divided by block
 */
__global__ void sgemm_naive(int N, int M, int K, float *a, float *b, float *c, float alpha, float beta)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if(x <= N && y <= K)
    {
        float tmp = 0.0f;
        for(int k = 0; k < M; k ++)
        {
            tmp += a[x * M + k] * b[k * K + y];
        }

        c[x * K + y] = alpha * tmp + beta;
    }
}
/**
 * Time:           0.524241ms.
 *  GFlop:          17.1841
 *  GFLOPS:         32.7789
 *  Percentage:     0.749233%.
 */
int main(int argc, char ** argv)
{
    if(argc < 2)
    {
        usage();
        exit(1);
    }

    N = atoi(argv[1]);
    size_t size = sizeof(float) * N * N;
    float flop = 1.0 * N * N * (2 * N + 1);
    
    float *h_a = new float[N * N];
    float *h_b = new float[N * N];
    float *h_c = new float[N * N];

    generate_float_matrix(h_a, N, N);
    generate_float_matrix(h_b, N, N);
    memset(h_c, 0, size);

    float *d_a;
    float *d_b;
    float *d_c;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);

    // fix typo.
    dim3 grid_dim = {N / BLK, N / BLK, 1};
    dim3 blk_dim = {BLK, BLK, 1};

    // start measuring.
    float elapsed; // in milisecond

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    sgemm_naive<<<grid_dim, blk_dim>>>(N, N, N, d_a, d_b, d_c, 1.0f, 0.0f);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    // std::cout << "GFLOPS: " << flops / 1000000000.0f / elapsed * 1000.0f << std::endl;
    
    float gflop = flop / 1000000000.0f;
    elapsed = elapsed / 1000.0f; // to second

    std::cout 
            << "Time:                                   \t" << elapsed << "ms.\n"
            << "GFlop:                                  \t" << gflop << "\n"
            << "GFLOPS:                                 \t" << gflop / elapsed << "\n"
            << "Percentage(compare to theoratical peak):\t" << (gflop / elapsed) / 4591.26f * 100.0 << "%.\n"
            << "Percentage(compare to cublas peak):     \t" << (gflop / elapsed) / 3737.3f * 100.0 << "%.\n";

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    return 0;
}