#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "common.cuh"
#include <assert.h>

void usage()
{
    std::cout << "Usage: ./a.out <matrix dimension>\n"
                << "Notice, the a x b = c\n"
                << "a: N x M matrix\n"
                << "b: M x K matrix\n"
                << "c: N x K matrix\n"
                // << "to align with cublas, we use b^T x a^T = c^T\n"
                // << "aka: b K x M matrix, a M x N matrix, c K x N matrix\n"
                << "In the example here the N = M = K\n";
}

/**
 * assuming all of data can be divided by block
 * completely doesnot understand this.
 */
template <
    const int BLK_N,
    const int BLK_M, // reduce the m to optimize blkm
    const int BLK_K,
    const int THREAD_N
    >
__global__ void sgemm_1d_tiling(int N, int M, int K, float *a, float *b, float *c, float alpha, float beta)
{
    assert(BLK_N * BLK_M == blockDim.x);
    assert(BLK_M * BLK_K == blockDim.x);
    int c_row = blockIdx.x;
    int c_col = blockIdx.y;

    int thread_row = threadIdx.x / BLK_K;
    int thread_col = threadIdx.x % BLK_K;

    int inner_a_row = threadIdx.x / BLK_M;
    int inner_a_col = threadIdx.x % BLK_M;

    int inner_b_row = threadIdx.x / BLK_K;
    int inner_b_col = threadIdx.x % BLK_K;

    a += c_row * BLK_N * M;
    b += c_col * BLK_K;
    c += c_row * K * BLK_N + c_col * BLK_K;

    __shared__ float a_share[BLK_N * BLK_M];
    __shared__ float b_share[BLK_M * BLK_K];

    float temp_arr[THREAD_N] = {0.0f};

    /**
     * A: N * M
     * B: M * K
     */
    for(int k = 0; k < M; k += BLK_M)
    {
        a_share[inner_a_row * BLK_M + inner_a_col] = a[inner_a_row * M + inner_a_col];
        b_share[inner_b_row * BLK_K + inner_b_col] = b[inner_b_row * K + inner_b_col];

        __syncthreads();

        a += BLK_M;
        b += BLK_M * K;

        for(int inner = 0; inner < BLK_M; inner ++)
        {
            // calculate each element of col of the tile per thread
            float temp_b = b_share[inner * BLK_K + thread_col];
            for(int tid = 0; tid < THREAD_N; tid++)
            {
                temp_arr[tid] += 
                    a_share[(thread_row * THREAD_N + tid) * BLK_M + inner]
                    * temp_b;
            }
        }

        __syncthreads();

    }

    for(int i = 0; i < THREAD_N; i ++)
        c[(thread_row * THREAD_N + i) * K + thread_col] = alpha * temp_arr[i] + beta;
}
/**
 * Time:                                           0.113557ms.
 * GFlop:                                          137.456
 * GFLOPS:                                         1210.46
 * Percentage(compare to theoratical peak):        26.3644%.
 * Percentage(compare to cublas peak):             31.7252%.
 */
int main(int argc, char ** argv)
{
    if(argc < 2)
    {
        usage();
        exit(1);
    }
    uint N;
    const int BLK_N = 64;
    const int BLK_M = 8;
    const int BLK_K = 64;
    const int THREAD_N = 8;

    N = atoi(argv[1]);

    size_t size = sizeof(float) * N * N;
    float flop = 1.0 * N * N * (2 * N + 1);
    
    float *h_a = new float[N * N];
    float *h_b = new float[N * N];
    float *h_c = new float[N * N];

    generate_float_matrix(h_a, N, N);
    generate_float_matrix(h_b, N, N);
    memset(h_c, 0, size);

    float *d_a;
    float *d_b;
    float *d_c;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);

    // fix typo.
    dim3 grid_dim = {N / BLK_N, N / BLK_K};
    dim3 blk_dim = {(BLK_N * BLK_K) / THREAD_N};

    // start measuring.
    float elapsed; // in milisecond

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    sgemm_1d_tiling<BLK_N, BLK_M, BLK_K, THREAD_N><<<grid_dim, blk_dim>>>(N, N, N, d_a, d_b, d_c, 1.0f, 0.0f);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    // std::cout << "GFLOPS: " << flops / 1000000000.0f / elapsed * 1000.0f << std::endl;

    float gflop = flop / 1000000000.0f;
    elapsed = elapsed / 1000.0f; // to second

    std::cout 
            << "Time:                                   \t" << elapsed << "ms.\n"
            << "GFlop:                                  \t" << gflop << "\n"
            << "GFLOPS:                                 \t" << gflop / elapsed << "\n"
            << "Percentage(compare to theoratical peak):\t" << (gflop / elapsed) / 4591.26f * 100.0 << "%.\n"
            << "Percentage(compare to cublas peak):     \t" << (gflop / elapsed) / 3737.3f * 100.0 << "%.\n";

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    return 0;
}