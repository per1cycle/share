
#include <hip/hip_runtime.h>
#include <iostream>
// #include <common.cuh>
int main()
{
    std::cout << "Cuda works" << std::endl;
    int M = 1024, N = 1024, K = 1024;

    float *h_a = (float*) malloc(sizeof(float) * M * K);
    float *h_b = (float*) malloc(sizeof(float) * K * N);
    float *h_c = (float*) malloc(sizeof(float) * M * N);

    float *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, M * K * sizeof(float));
    hipMalloc((void **)&d_b, K * N * sizeof(float));
    hipMalloc((void **)&d_c, M * N * sizeof(float));

    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    ///////////////////////////////////////////////////////////////////////////////////////
    // run kernel here

    ///////////////////////////////////////////////////////////////////////////////////////
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);
    std::cout << "Cuda finished" << std::endl;
}