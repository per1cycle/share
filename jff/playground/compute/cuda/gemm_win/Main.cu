#include "Kernels.cuh"
#include <iostream>

#include <hip/hip_runtime.h>

int main(int argc, char ** argv)
{
	if (argc != 3)
	{
		std::cerr << "Usage: " << argv[0] << " <mode> <N> " << std::endl;
		return -1;
	}

	unsigned int N = atoi(argv[2]);
	int mode = atoi(argv[1]);
	std::cout << "Running with mode: " << mode << std::endl;
	unsigned int size = N * N;

	float* h_a, * h_b, * h_c;

	h_a = (float*)malloc(size * sizeof(float));
	h_b = (float*)malloc(size * sizeof(float));
	h_c = (float*)malloc(size * sizeof(float));

	generate_float_matrix(h_a, N, N);
	generate_float_matrix(h_b, N, N);

	float* d_a, * d_b, * d_c;
	hipMalloc((void**)&d_a, size * sizeof(float));
	hipMalloc((void**)&d_b, size * sizeof(float));
	hipMalloc((void**)&d_c, size * sizeof(float));

	hipMemcpy(d_a, h_a, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, size * sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	float elapsed = 0.0f;
    
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	run_kernel(mode, N, N, N, d_a, d_b, d_c, 1.0f, 0.0f);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);

	// show result
	float flop = 1.0 * N * N * (2 * N + 1);
	float gflop = flop / 1000000000.0f;
	elapsed = elapsed / 1000.0f; // to second

	std::cout
		<< "Time:                                   \t" << elapsed << " s.\n"
		<< "GFlop:                                  \t" << gflop << "\n"
		<< "GFLOPS:                                 \t" << gflop / elapsed << "\n";

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}