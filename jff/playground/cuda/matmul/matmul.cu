
#include <hip/hip_runtime.h>
#include <iostream>

const int N = 1024;
__global__ void MatAdd(float A[N], float B[N], float C[N])
{
    int i = threadIdx.x;
    C[i] = A[i] + B[N - i - 1];
}

int main()
{
    std::cout << "Cuda works" << std::endl;
    float A[N], B[N], C[N];
    for (int i = 0; i < N; i++)
    {
        A[i] = i;
        B[i] = i;
    }
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N * sizeof(float), hipMemcpyHostToDevice);
    MatAdd<<<4, N / 4>>>(d_A, d_B, d_C);
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Cuda finished" << std::endl;
}