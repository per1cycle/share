
#include <hip/hip_runtime.h>
#include <iostream>


// device code
__global__ void PitchedInit(float* pitchPtr, size_t pitch, int width, int height)
{
    for(int r = 0; r < height; r ++)
    {
        float* row = (float*)((char*)pitchPtr + r * pitch);
        for(int c = 0; c < width; c ++)
        {
            row[c] = 1;
        }
    }
}
int main(int argc, char const *argv[])
{
    const int w = 4096, h = 4096;
    float* pitchPtr;
    size_t pitch = 0;

    float* result = new float[w * h * sizeof(float)];

    // https://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
    hipError_t err = hipMallocPitch(&pitchPtr, &pitch, w * sizeof(float), h);
    if(err != hipSuccess) std::cout << hipGetErrorString(err) << '\n';

    PitchedInit<<<100, 128>>>(pitchPtr, pitch, w, h);
    hipMemcpy2D(result, w * sizeof(float), pitchPtr, pitch, w * sizeof(float), h, hipMemcpyDeviceToHost);

    for(int i = 0; i < w * h; i ++)
    {
        std::cout << result[i] << ' ';
        if(i && i % w == 0) std::cout << '\n';
    }

    std::cout << pitch << '\n';

    hipFree(pitchPtr);
    free(result);

    return 0;
}
