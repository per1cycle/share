
#include <hip/hip_runtime.h>
#include <iostream>


// device code
__global__ void pitch_init(float* pitch_ptr, size_t pitch, int width, int height)
{
    for(int r = 0; r < height; r ++)
    {
        float* row = (float*)((char*)pitch_ptr + r * pitch);
        for(int c = 0; c < width; c ++)
        {
            row[c] = 1;
        }
    }
}

int main(int argc, char const *argv[])
{
    const int w = 4096, h = 4096;
    float* pitch_ptr;
    size_t pitch = 0;

    float* result = new float[w * h * sizeof(float)];

    // https://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
    hipError_t err = hipMallocPitch(&pitch_ptr, &pitch, w * sizeof(float), h);
    if(err != hipSuccess) std::cout << hipGetErrorString(err) << '\n';

    pitch_init<<<100, 128>>>(pitch_ptr, pitch, w, h);
    hipMemcpy2D(result, w * sizeof(float), pitch_ptr, pitch, w * sizeof(float), h, hipMemcpyDeviceToHost);

    for(int i = 0; i < w * h; i ++)
    {
        std::cout << result[i] << ' ';
        if(i && i % w == 0) std::cout << '\n';
    }

    std::cout << pitch << '\n';

    hipFree(pitch_ptr);
    free(result);

    return 0;
}
