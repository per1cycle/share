#include "hip/hip_runtime.h"
#include <random>
#include "common/utils.h"
const int N = 4096;
const int blocks_per_grid = 100, threads_per_block = 1024;

// accelerate 1d array operation
__global__ void accelerate_1d_arr(float *arr_1d_a, float *arr_1d_b, float *arr_1d_c)
{
    int idx = threadIdx.x;
    arr_1d_c[idx] = arr_1d_a[idx] + arr_1d_b[idx];
}

// accelerate 2d operation
__global__ void accelerate_2d_array(float *DevPtrA, float *DevPtrB, float *DevPtrC, size_t Pitch, int Width, int Height)
{

}

int main()
{
    
    float *host_arr_1d_a;
    float *host_arr_1d_b;
    float *host_arr_1d_c;

    host_arr_1d_a = new float[N];
    host_arr_1d_b = new float[N];
    host_arr_1d_c = new float[N];

    float *device_arr_1d_a;
    float *device_arr_1d_b;
    float *device_arr_1d_c;

    hipMalloc(&device_arr_1d_a, sizeof(float) * N);
    hipMalloc(&device_arr_1d_b, sizeof(float) * N);
    hipMalloc(&device_arr_1d_c, sizeof(float) * N);

    hipMemcpy(device_arr_1d_a, host_arr_1d_a, N, hipMemcpyHostToDevice);
    hipMemcpy(device_arr_1d_b, host_arr_1d_b, N, hipMemcpyHostToDevice);
    hipMemcpy(device_arr_1d_c, host_arr_1d_c, N, hipMemcpyHostToDevice);
    
    accelerate_1d_arr<<<blocks_per_grid, threads_per_block>>>(device_arr_1d_a, device_arr_1d_b, device_arr_1d_c);

}