#include <iostream>
#include "hip/hip_runtime.h"


int main()
{
    int device_count = 0;
    hipGetDeviceCount(&device_count);
    std::cout << "GPUs: " << device_count << std::endl;

    for(int i = 0; i < device_count; i ++)
    {
        int dev;
        hipDeviceProp_t prop;
        hipError_t err = hipGetDevice(&dev);
        hipGetDeviceProperties(&prop, dev);
        std::cout << "Gpu name: " << prop.name << std::endl;
    }

    return 0;
}
